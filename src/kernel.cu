#include "hip/hip_runtime.h"
#include "../include/mtx_utils.h"
#include "../include/test_utils.h"

#ifndef SELECT_KERNEL
#error "Please define SELECT_KERNEL to select the kernel to use."
#endif

#if SELECT_KERNEL == 1
  #define KERNEL SpMV_OneThreadPerRow
  #define KERNEL_NAME "SpMV_OneThreadPerRow"
  #define KERNEL_PARAMS matrix.nrows, matrix.row_ptr, matrix.col_idx, matrix.values, vec, result
#elif SELECT_KERNEL == 2
  #define KERNEL SpMV_OneWarpPerRow
  #define KERNEL_NAME "SpMV_OneWarpPerRow"
  #define KERNEL_PARAMS matrix.nrows, matrix.row_ptr, matrix.col_idx, matrix.values, vec, result
#elif SELECT_KERNEL == 3
  #define KERNEL SpMV_coalescedBins
  #define KERNEL_NAME "SpMV_coalescedBins"
  #define KERNEL_PARAMS \
    matrix.nrows, matrix.row_ptr, matrix.col_idx, matrix.values, vec, result, bin_rows
#elif SELECT_KERNEL == 4
  #define KERNEL SpMV_Hybrid
  #define KERNEL_NAME "SpMV_Hybrid"
  #define KERNEL_PARAMS                                                                   \
    matrix.nrows, matrix.row_ptr, matrix.col_idx, matrix.values, vec, result, short_rows, \
    long_rows, num_short_rows, num_long_rows
#endif

#define CUDA_CHECK_KERNEL()                                         \
  do {                                                              \
    hipError_t err = hipGetLastError();                           \
    if (err != hipSuccess) {                                       \
      fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
      return 1;                                                     \
    }                                                               \
  } while (0)

#define BLOCK_DIM 256
#define WARP_SIZE 32

// ******************************
// *     Kernel Definitions     *
// ******************************

__global__ void SpMV_OneThreadPerRow(const int rows, const int *row_ptr, const int *col_idx,
                                     const double *values, const double *vec, double *result) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < rows) {
    double sum = 0.0;
    int start = row_ptr[row];
    int end = row_ptr[row + 1];
    for (int j = start; j < end; j++) {
      sum += values[j] * __ldg(&vec[col_idx[j]]);
    }
    result[row] = sum;
  }
}

__global__ void SpMV_OneWarpPerRow(const int rows, const int *row_ptr, const int *col_idx,
                                   const double *values, const double *vec, double *result) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int warp_id = tid / WARP_SIZE;
  int lane_id = tid % WARP_SIZE;

  if (warp_id < rows) {
    double sum = 0.0;
    int start = row_ptr[warp_id];
    int end = row_ptr[warp_id + 1];
    for (int j = start + lane_id; j < end; j += WARP_SIZE) {
      sum += values[j] * __ldg(&vec[col_idx[j]]);
    }

    for (int offset = 16; offset > 0; offset /= 2) {
      sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }
    if (lane_id == 0) {
      result[warp_id] = sum;
    }
  }
}

__global__ void SpMV_coalescedBins(const int num_bins, const int *row_ptr, const int *col_idx,
                                   const double *values, const double *vec, double *result,
                                   const int *bin_rows) {
  int warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
  int lane_id = threadIdx.x % WARP_SIZE;

  if (warp_id >= num_bins) return;

  int row_start = bin_rows[warp_id];
  int row_end = bin_rows[warp_id + 1];

  int nnz_start = row_ptr[row_start];
  int nnz_end = row_ptr[row_end];
  int total_nnz = nnz_end - nnz_start;

  for (int i = lane_id; i < total_nnz; i += WARP_SIZE) {
    int idx = nnz_start + i;
    int row = row_start;
    while (idx >= row_ptr[row + 1] && row < row_end - 1) {
      row++;
    }

    double val = values[idx] * __ldg(&vec[col_idx[idx]]);
    atomicAdd(&result[row], val);
  }
}

__global__ void SpMV_Hybrid(const int rows, const int *row_ptr, const int *col_idx,
                            const double *values, const double *vec, double *result,
                            const int *short_rows, const int *long_rows, int num_short_rows,
                            int num_long_rows) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < num_short_rows) {
    int row = short_rows[tid];
    int start = row_ptr[row];
    int end = row_ptr[row + 1];
    double sum = 0.0;

    for (int j = start; j < end; j++) {
      sum += values[j] * __ldg(&vec[col_idx[j]]);
    }
    result[row] = sum;
  } else {
    tid = tid - num_short_rows;
    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;

    if (warp_id < num_long_rows) {
      int row = long_rows[warp_id];
      int start = row_ptr[row];
      int end = row_ptr[row + 1];
      double sum = 0.0;

      for (int j = start + lane_id; j < end; j += WARP_SIZE) {
        sum += values[j] * __ldg(&vec[col_idx[j]]);
      }

      for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
      }
      if (lane_id == 0) {
        result[row] = sum;
      }
    }
  }
}

// ******************************
// *       Main Function        *
// ******************************

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <matrix_file>\n", argv[0]);
    return -1;
  }

  int device = -1;
  hipGetDevice(&device);

  if (device == -1) {
    fprintf(stderr, "Error: No GPU device found.\n");
    return 1;
  }

  double finalResult = 0.0;
  float times[REP] = {0};
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Matrix, vector, and result vector
  COO_matrix coo_matrix;
  CSR_matrix matrix;
  double *vec = NULL;
  double *result = NULL;

  // Read the matrix from file
  read_COO_mtx(argv[1], &coo_matrix);

  // Allocate memory for the CSR matrix and vectors
  matrix.nrows = coo_matrix.nrows;
  matrix.ncols = coo_matrix.ncols;
  matrix.nnz = coo_matrix.nnz;
  hipMallocManaged(&matrix.row_ptr, (matrix.nrows + 1) * sizeof(int));
  hipMallocManaged(&matrix.col_idx, matrix.nnz * sizeof(int));
  hipMallocManaged(&matrix.values, matrix.nnz * sizeof(double));
  hipMallocManaged(&vec, matrix.ncols * sizeof(double));
  hipMallocManaged(&result, matrix.nrows * sizeof(double));

  if (!matrix.row_ptr || !matrix.col_idx || !matrix.values || !vec || !result) {
    fprintf(stderr, "Error: Failed to allocate memory for matrix or vector.\n");
    free(coo_matrix.row_idx);
    free(coo_matrix.col_idx);
    free(coo_matrix.values);
    hipFree(matrix.row_ptr);
    hipFree(matrix.col_idx);
    hipFree(matrix.values);
    hipFree(vec);
    hipFree(result);
    return 1;
  }

  // Convert COO to CSR format and initialize the vector
  COO_to_CSR(&coo_matrix, &matrix);
  free(coo_matrix.row_idx);
  free(coo_matrix.col_idx);
  free(coo_matrix.values);
  init_RandVector(vec, matrix.ncols);

  // Prefetch data to the GPU
  hipMemPrefetchAsync(matrix.row_ptr, (matrix.nrows + 1) * sizeof(int), device);
  hipMemPrefetchAsync(matrix.col_idx, matrix.nnz * sizeof(int), device);
  hipMemPrefetchAsync(matrix.values, matrix.nnz * sizeof(double), device);
  hipMemPrefetchAsync(vec, matrix.ncols * sizeof(double), device);
  hipMemPrefetchAsync(result, matrix.nrows * sizeof(double), device);
  hipDeviceSynchronize();

  // Based on the selected kernel, set the number of blocks and threads
  int gridDim = 0;
  #if SELECT_KERNEL == 1
    gridDim = (matrix.nrows + BLOCK_DIM - 1) / BLOCK_DIM;

  #elif SELECT_KERNEL == 2
    gridDim = ((matrix.nrows * WARP_SIZE) + BLOCK_DIM - 1) / BLOCK_DIM;

  #elif SELECT_KERNEL == 3
    int *host_bin_rows = (int *)malloc((matrix.nrows + 1) * sizeof(int));
    int num_bins =
        build_coalesced_row_bins(matrix.row_ptr, matrix.nrows, host_bin_rows, WARP_SIZE);

    int *bin_rows = NULL;
    hipMallocManaged(&bin_rows, (num_bins + 1) * sizeof(int));
    hipMemcpy(bin_rows, host_bin_rows, (num_bins + 1) * sizeof(int), hipMemcpyHostToDevice);
    free(host_bin_rows);

    hipMemPrefetchAsync(bin_rows, (num_bins + 1) * sizeof(int), device);
    hipDeviceSynchronize();

    gridDim = ((num_bins * WARP_SIZE) + BLOCK_DIM - 1) / BLOCK_DIM;

  #elif SELECT_KERNEL == 4
    int *host_short_rows = (int *)malloc(matrix.nrows * sizeof(int));
    int *host_long_rows = (int *)malloc(matrix.nrows * sizeof(int));
    int num_short_rows = 0, num_long_rows = 0;

    classify_rows(matrix.row_ptr, matrix.nrows,
                  host_short_rows, host_long_rows,
                  &num_short_rows, &num_long_rows,
                  WARP_SIZE * 2);

    int *short_rows = NULL, *long_rows = NULL;

    hipMallocManaged(&short_rows, (num_short_rows > 0 ? num_short_rows : 1) * sizeof(int));
    hipMemcpy(short_rows, host_short_rows, num_short_rows * sizeof(int), hipMemcpyHostToDevice);

    hipMallocManaged(&long_rows, (num_long_rows > 0 ? num_long_rows : 1) * sizeof(int));
    hipMemcpy(long_rows, host_long_rows, num_long_rows * sizeof(int), hipMemcpyHostToDevice);

    free(host_short_rows);
    free(host_long_rows);

    if (num_short_rows > 0) {
      hipMemPrefetchAsync(short_rows, num_short_rows * sizeof(int), device);
    }
    if (num_long_rows > 0) {
      hipMemPrefetchAsync(long_rows, num_long_rows * sizeof(int), device);
    }
    hipDeviceSynchronize();

    int total_threads = num_short_rows + num_long_rows * WARP_SIZE;
    gridDim = (total_threads + BLOCK_DIM - 1) / BLOCK_DIM;
  #endif

  printf("Using kernel: \t\t%s\n", KERNEL_NAME);
  printf("Using matrix: \t\t%s\n\n", argv[1]);
  print_mtx_stats(&matrix);
  fflush(stdout);

  // Warm up
  for (int i = 0; i < WARM_UP; i++) {
    KERNEL<<<gridDim, BLOCK_DIM>>>(KERNEL_PARAMS);
    hipDeviceSynchronize();

    CUDA_CHECK_KERNEL();

    if (i == 0) {
      for (int j = 0; j < matrix.nrows; j++) {
        finalResult += result[j];
      }
    }
  }

  // Benchmarking phase
  for (int i = 0; i < REP; i++) {
    hipEventRecord(start);

    KERNEL<<<gridDim, BLOCK_DIM>>>(KERNEL_PARAMS);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    CUDA_CHECK_KERNEL();

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    times[i] = ms * 1e-3;  // Convert to seconds
  }

  // Print results
  float meanTime = arithmetic_mean(times, REP);
  float flopCount = 2.0 * matrix.nnz;
  float gflops = calculate_GFlops(flopCount, meanTime);

  size_t readedBytes = matrix.nnz * (sizeof(int) + sizeof(double)) +  // col_idx and values
                       (matrix.nrows + 1) * sizeof(int) +             // row_ptr
                       matrix.ncols * sizeof(double);                 // vec

  #if SELECT_KERNEL == 3
    readedBytes += (matrix.nrows + 1) * sizeof(int);  // bin_rows
  #elif SELECT_KERNEL == 4
    readedBytes += num_short_rows * sizeof(int);  // short_rows
    readedBytes += num_long_rows * sizeof(int);   // long_rows
  #endif

  size_t writtenBytes = matrix.nrows * sizeof(double);  // result
  size_t totalBytes = readedBytes + writtenBytes;
  float bandwidth = (float)totalBytes / (meanTime * 1e9);  // GB/s

  printf("Sum of resulting vector: %f\n", finalResult);
  printf("Mean time: %f ms\n", meanTime * 1e3);
  printf("GFlops: %f\n", gflops);
  printf("Bandwidth: %f GB/s\n", bandwidth);

  // Free memory
  hipFree(matrix.row_ptr);
  hipFree(matrix.col_idx);
  hipFree(matrix.values);
  hipFree(vec);
  hipFree(result);
  hipDeviceReset();
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}