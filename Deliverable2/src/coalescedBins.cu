#include "hip/hip_runtime.h"
#include "../include/mtx_utils.h"
#include "../include/test_utils.h"

#define KERNEL SpMV_coalescedBins
#define KERNEL_NAME "SpMV_coalescedBins"
#define KERNEL_PARAMS \
  num_bins, matrix.row_ptr, matrix.col_idx, matrix.values, vec, result, bin_rows

// ===================== Helpers & safety =====================
#define CUDA_CHECK(stmt)                                            \
  do {                                                              \
    hipError_t _err = (stmt);                                      \
    if (_err != hipSuccess) {                                      \
      fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(_err));                            \
      return 1;                                                     \
    }                                                               \
  } while (0)

#define CUDA_CHECK_KERNEL()                                           \
  do {                                                                \
    hipError_t err = hipGetLastError();                             \
    if (err != hipSuccess) {                                         \
      fprintf(stderr, "Kernel error: %s\n", hipGetErrorString(err)); \
      return 1;                                                       \
    }                                                                 \
  } while (0)

#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

#ifndef BLOCK_DIM
#define BLOCK_DIM 256
#endif

// ===================== Kernel =====================
__global__ void SpMV_coalescedBins(const int num_bins, const int *__restrict__ row_ptr,
                                   const int *__restrict__ col_idx,
                                   const double *__restrict__ values,
                                   const double *__restrict__ vec, double *__restrict__ result,
                                   const int *__restrict__ bin_rows) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int lane = threadIdx.x & (WARP_SIZE - 1);
  const int total_threads = gridDim.x * blockDim.x;
  const int warps_per_grid = total_threads / WARP_SIZE;
  int base_bw = tid / WARP_SIZE;

  for (int bw = base_bw; bw < num_bins; bw += warps_per_grid) {
    const int row_start = __ldg(&bin_rows[bw]);
    const int row_end = __ldg(&bin_rows[bw + 1]);

    for (int row = row_start; row < row_end; ++row) {
      const int start = __ldg(&row_ptr[row]);
      const int end = __ldg(&row_ptr[row + 1]);
      double sum = 0.0;

#pragma unroll
      for (int j = start + lane; j < end; j += WARP_SIZE) {
        double val = __ldg(&values[j]);
        int col = __ldg(&col_idx[j]);
        sum += val * __ldg(&vec[col]);
      }

      for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
      }

      if (lane == 0) {
        result[row] = sum;
      }
    }
  }
}

// ===================== Main =====================
int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Usage: %s <matrix_file>\n", argv[0]);
    return -1;
  }

  int device = 0;
  CUDA_CHECK(hipGetDevice(&device));

  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));
  const int SM = prop.multiProcessorCount;

  double finalResult = 0.0;
  float times[REP] = {0.0f};
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  // Matrix, vector, and result vector
  COO_matrix coo_matrix{};
  CSR_matrix matrix{};
  double *vec = nullptr;
  double *result = nullptr;

  // Matrix statistics
  int max_nnz = 0;
  int min_nnz = INT_MAX;
  double avg_nnzPerRow = 0.0;

  // Read the matrix from file
  read_COO_mtx(argv[1], &coo_matrix);

  // Allocate memory for the CSR matrix and vectors
  matrix.nrows = coo_matrix.nrows;
  matrix.ncols = coo_matrix.ncols;
  matrix.nnz = coo_matrix.nnz;
  CUDA_CHECK(hipMallocManaged(&matrix.row_ptr, (matrix.nrows + 1) * sizeof(int)));
  CUDA_CHECK(hipMallocManaged(&matrix.col_idx, matrix.nnz * sizeof(int)));
  CUDA_CHECK(hipMallocManaged(&matrix.values, matrix.nnz * sizeof(double)));
  CUDA_CHECK(hipMallocManaged(&vec, matrix.ncols * sizeof(double)));
  CUDA_CHECK(hipMallocManaged(&result, matrix.nrows * sizeof(double)));

  // Convert COO to CSR format and initialize the vector
  COO_to_CSR(&coo_matrix, &matrix);
  free(coo_matrix.row_idx);
  free(coo_matrix.col_idx);
  free(coo_matrix.values);
  init_RandVector(vec, matrix.ncols);

  // Advise CUDA on memory usage
  CUDA_CHECK(hipMemAdvise(matrix.values, matrix.nnz * sizeof(double),
                           hipMemAdviseSetPreferredLocation, device));
  CUDA_CHECK(hipMemAdvise(matrix.col_idx, matrix.nnz * sizeof(int),
                           hipMemAdviseSetPreferredLocation, device));
  CUDA_CHECK(hipMemAdvise(matrix.row_ptr, (matrix.nrows + 1) * sizeof(int),
                           hipMemAdviseSetReadMostly, device));
  CUDA_CHECK(
      hipMemAdvise(vec, matrix.ncols * sizeof(double), hipMemAdviseSetReadMostly, device));

  // Prefetch data to the GPU
  CUDA_CHECK(hipMemPrefetchAsync(matrix.row_ptr, (matrix.nrows + 1) * sizeof(int), device));
  CUDA_CHECK(hipMemPrefetchAsync(matrix.col_idx, matrix.nnz * sizeof(int), device));
  CUDA_CHECK(hipMemPrefetchAsync(matrix.values, matrix.nnz * sizeof(double), device));
  CUDA_CHECK(hipMemPrefetchAsync(vec, matrix.ncols * sizeof(double), device));
  CUDA_CHECK(hipMemPrefetchAsync(result, matrix.nrows * sizeof(double), device));
  CUDA_CHECK(hipDeviceSynchronize());

  // Set the number of blocks and threads
  int gridDim = 0;
  int *host_bin_rows = (int *)malloc((matrix.nrows + 1) * sizeof(int));
  int num_bins =
      build_coalesced_row_bins(matrix.row_ptr, matrix.nrows, host_bin_rows, WARP_SIZE);
  int *bin_rows = nullptr;
  CUDA_CHECK(hipMallocManaged(&bin_rows, (num_bins + 1) * sizeof(int)));
  CUDA_CHECK(hipMemcpy(bin_rows, host_bin_rows, (num_bins + 1) * sizeof(int),
                        hipMemcpyHostToDevice));
  free(host_bin_rows);
  CUDA_CHECK(hipMemPrefetchAsync(bin_rows, (num_bins + 1) * sizeof(int), device));
  CUDA_CHECK(hipDeviceSynchronize());

  const long long total_warps = num_bins;
  const long long total_threads = total_warps * WARP_SIZE;
  gridDim =
      (int)MIN((total_threads + BLOCK_DIM - 1) / (long long)BLOCK_DIM, (long long)SM * 8);

  printf("Using kernel: \t\t%s\n", KERNEL_NAME);
  printf("Using matrix: \t\t%s\n", argv[1]);
  printf("Using block size: \t%d\n\n", BLOCK_DIM);
  print_mtx_stats(&matrix, &max_nnz, &min_nnz, &avg_nnzPerRow);
  fflush(stdout);

  for (int i = 0; i < WARM_UP; ++i) {
    KERNEL<<<gridDim, BLOCK_DIM>>>(KERNEL_PARAMS);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK_KERNEL();
  }
  for (int i = 0; i < REP; ++i) {
    CUDA_CHECK(hipEventRecord(start));
    KERNEL<<<gridDim, BLOCK_DIM>>>(KERNEL_PARAMS);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK_KERNEL();
    float ms = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    times[i] = ms * 1e-3f;
  }

  // Calculate results
  const float meanTime = arithmetic_mean(times, REP);  // sec
  const float flopCount = 2.0f * (float)matrix.nnz;
  const float gflops = calculate_GFlops(flopCount, meanTime);

  // Bandwidth calculation based on the worst-case scenario
  const int Bd = sizeof(double);  // 8 bytes
  const int Bi = sizeof(int);     // 4 bytes

  size_t readBytes = (size_t)matrix.nrows * (Bi + Bi)       // row_ptr[row], row_ptr[row+1]
                     + (size_t)matrix.nnz * (Bi + Bd + Bi)  // col_idx, values, vec
                     + (size_t)(num_bins + 1) * Bi;         // bin_rows
  size_t writeBytes = (size_t)matrix.nrows * Bd;            // result[row]

  const size_t totalBytes = readBytes + writeBytes;
  const float bandwidthGBs = (float)totalBytes / (meanTime * 1e9f);

  // Copy result
  CUDA_CHECK(hipMemPrefetchAsync(result, matrix.nrows * sizeof(double), hipCpuDeviceId));
  CUDA_CHECK(hipDeviceSynchronize());
  for (int r = 0; r < matrix.nrows; ++r) finalResult += result[r];

  printf("Sum of resulting vector: %f\n", finalResult);
  printf("Mean time: %f ms\n", meanTime * 1e3f);
  printf("GFlops: %f\n", gflops);
  printf("Bandwidth: %f GB/s\n", bandwidthGBs);

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  CUDA_CHECK(hipFree(matrix.row_ptr));
  CUDA_CHECK(hipFree(matrix.col_idx));
  CUDA_CHECK(hipFree(matrix.values));
  CUDA_CHECK(hipFree(vec));
  CUDA_CHECK(hipFree(result));
  CUDA_CHECK(hipDeviceReset());

  return 0;
}
